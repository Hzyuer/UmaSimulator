#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdint>

// 非原位ReLU核函数
__global__ void reluOutOfPlaceKernel(const float* input, float* output, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    output[i] = max(0.0f, input[i]);
  }
}

// 调用非原位ReLU核函数的接口
hipError_t reluOutOfPlace(const float* input, float* output, int n) {
  int blockSize = 512;
  int numBlocks = (n + blockSize - 1) / blockSize;
  reluOutOfPlaceKernel << <numBlocks, blockSize >> > (input, output, n);
  hipDeviceSynchronize();
  return hipGetLastError();
}

// 原位ReLU核函数
__global__ void reluInPlaceKernel(float* data, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    data[i] = max(0.0f, data[i]);
  }
}

// 调用原位ReLU核函数的接口
hipError_t reluInPlace(float* data, int n) {
  int blockSize = 512;
  int numBlocks = (n + blockSize - 1) / blockSize;
  reluInPlaceKernel << <numBlocks, blockSize >> > (data, n);
  hipDeviceSynchronize();
  return hipGetLastError();
}
/*
//裁剪矩阵，batchSize列c行，列优先
__global__ void cropMatrixRowsKernel(float* d_out, const float* d_in, int c, int batchSize, int start, int end) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y + start;  // 起始于第a行

  if (col < start && row < end) {
    d_out[col * (end - start) + (row - start)] = d_in[col * c + row];
  }
}
void cropMatrixRowsCUDA(float* d_out, const float* d_in, int c, int batchSize, int start, int end) {
  // 检查a和b的合法性，确保它们在矩阵范围内
  if (start < 0 || end >= c || start > end) {
    return; // 可能需要一些错误处理
  }

  int newRowSize = end - start;

  dim3 blockSize(32, 16); // 根据实际情况调整
  dim3 gridSize((batchSize + blockSize.x - 1) / blockSize.x, (newRowSize + blockSize.y - 1) / blockSize.y);

  cropMatrixRowsKernel <<<gridSize, blockSize>>> (d_out, d_in, c, batchSize, start, end);

  // 调用者应检查CUDA调用后的错误
}
*/
__global__ void cropMatrixRowsKernel(float* d_out, const float* d_in, int batchsize, int c, int start, int end) {
  int batchIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int fIdx = blockIdx.y * blockDim.y + threadIdx.y + start;  // 从第start行开始

  if (fIdx >= start && fIdx < end) {
    // 计算源矩阵和目标矩阵中的索引
    int idx_in = fIdx + batchIdx * c;               // 源矩阵列优先索引
    int idx_out = (fIdx - start) + batchIdx * (end - start);  // 输出矩阵列优先索引
    d_out[idx_out] = d_in[idx_in];
  }
}
hipError_t cropMatrixRowsCUDA(float* d_out, const float* d_in, int batchsize, int c, int start, int end) {
  // 检查start和end的合法性
  if (start < 0 || end > c || start >= end) {
    return hipErrorInvalidValue; // 错误处理
  }

  int cropSize = end - start;  // 裁剪区域的行数

  // 定义每个块的线程维度
  dim3 blockSize(32, 16);

  // 计算所需的块数来覆盖矩阵
  dim3 gridSize((batchsize + blockSize.x - 1) / blockSize.x, (cropSize + blockSize.y - 1) / blockSize.y);

  // 启动内核
  cropMatrixRowsKernel << <gridSize, blockSize >> > (d_out, d_in, batchsize, c, start, end);

  // 等待 GPU 完成
  hipDeviceSynchronize();

  return hipGetLastError();
}

__global__ void addThreeFeaturesKernel(float* A, float* B, float* C, int batchsize, int N_Person, int N_Card, int encoderC) {
  int encoderIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int personIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int batchIdx = blockIdx.z * blockDim.z + threadIdx.z;

  if (encoderIdx < encoderC && personIdx < N_Person && batchIdx < batchsize) {
    // 对于列优先存储，encoderC变化最快
    int idxC = encoderIdx + personIdx * encoderC + batchIdx * encoderC * N_Person;
    int idxA = encoderIdx + batchIdx * encoderC;  // A的索引

    // 对C[:,:,:] += A.unsqueeze(1)操作
    C[idxC] += A[idxA];

    // 对C[:,0:N_Card,:] += B操作
    if (personIdx < N_Card) {
      int idxB = encoderIdx + personIdx * encoderC + batchIdx * encoderC * N_Card;
      C[idxC] += B[idxB];
    }
  }
}
hipError_t addThreeFeatures(float* A, float* B, float* C, int batchsize, int N_Person, int N_Card, int encoderC) {
  // 定义block和grid的大小
  dim3 blockSize(16, 1, 32);
  dim3 gridSize((encoderC + blockSize.x - 1) / blockSize.x,
    (N_Person + blockSize.y - 1) / blockSize.y,
    (batchsize + blockSize.z - 1) / blockSize.z);

  // 调用核函数
  addThreeFeaturesKernel << <gridSize, blockSize >> > (A, B, C, batchsize, N_Person, N_Card, encoderC);

  // 检查CUDA错误和同步
  hipDeviceSynchronize();
  return hipGetLastError();
}

__global__ void broadcastDim1AddKernel(float* target, float* x, int batchsize, int N_Person, int encoderC) {
  int encoderIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int personIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int batchIdx = blockIdx.z * blockDim.z + threadIdx.z;

  if (encoderIdx < encoderC && personIdx < N_Person && batchIdx < batchsize) {
    // 对于列优先存储，encoderC变化最快
    int idxC = encoderIdx + personIdx * encoderC + batchIdx * encoderC * N_Person;
    int idxA = encoderIdx + batchIdx * encoderC;  // A的索引

    // 对C[:,:,:] += A.unsqueeze(1)操作
    target[idxC] += x[idxA];

  }
}
hipError_t broadcastDim1Add(float* target, float* x, int batchsize, int N_Person, int encoderC) {
  // 定义block和grid的大小
  dim3 blockSize(16, 1, 32);
  dim3 gridSize((encoderC + blockSize.x - 1) / blockSize.x,
    (N_Person + blockSize.y - 1) / blockSize.y,
    (batchsize + blockSize.z - 1) / blockSize.z);

  // 调用核函数
  broadcastDim1AddKernel << <gridSize, blockSize >> > (target, x, batchsize, N_Person, encoderC);

  // 检查CUDA错误和同步
  hipDeviceSynchronize();
  return hipGetLastError();
}

__global__ void addInPlaceKernel(float* A, const float* B, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < numElements) {
    A[i] += B[i];
  }
}

hipError_t addInPlace(float* A, const float* B, int numElements) {
  int blockSize = 512;
  int numBlocks = (numElements + blockSize - 1) / blockSize;
  addInPlaceKernel << <numBlocks, blockSize >> > (A, B, numElements);

  // 检查CUDA错误和同步
  hipDeviceSynchronize();
  return hipGetLastError();
}

__global__ void sumDim1Kernel(float* output, const float* input, int batchSize, int dim1Size, int channel) {
  int batch = blockIdx.x * blockDim.x + threadIdx.x;
  int ch = blockIdx.y * blockDim.y + threadIdx.y;

  if (batch < batchSize && ch < channel) {
    float sum = 0.0f;
    for (int i = 0; i < dim1Size; ++i) {
      sum += input[(batch * dim1Size * channel) + (i * channel) + ch];
    }
    output[batch * channel + ch] = sum;
  }
}

hipError_t sumDim1(float* output, float* input, int batchSize, int dim1Size, int channel) {
  dim3 blockDim(32, 16);
  dim3 gridDim((batchSize + blockDim.x - 1) / blockDim.x, (channel + blockDim.y - 1) / blockDim.y);

  sumDim1Kernel << <gridDim, blockDim >> > (output, input, batchSize, dim1Size, channel);

  // 检查是否有任何错误发生
  hipDeviceSynchronize();
  return hipGetLastError();
}

__global__ void sparseToDenseKernel(uint32_t* idx, float* value, float* output, int m) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < m) {
    output[idx[index]] = value[index];
  }
}

hipError_t sparseToDense(uint32_t* idx, float* value, float* output, int m) {
  // 假设output已经分配并初始化为0

  // 设置线程块和网格大小
  int threadsPerBlock = 512;
  int blocksPerGrid = (m + threadsPerBlock - 1) / threadsPerBlock;

  // 调用核函数
  sparseToDenseKernel << <blocksPerGrid, threadsPerBlock >> > (idx, value, output, m);

  // 等待GPU完成
  hipDeviceSynchronize();
  return hipGetLastError();
}

const int NNINPUT_MAX_FLOAT = 192; //nninput里面最多有多少个非0非1的数
const int NNINPUT_MAX_ONES = 192; //nninput里面最多有多少个1
__global__ void decompressNNInputKernel(uint16_t* onesIdx, uint16_t* floatIdx, float* floatValue, float* output, int m, int nninputSize) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < m) {
    for (int i = 0; i < NNINPUT_MAX_FLOAT; ++i) {
      int fIdx = NNINPUT_MAX_FLOAT * index + i;
      int cIdx = floatIdx[fIdx];
      if (cIdx < nninputSize)
        output[nninputSize * index + cIdx] = floatValue[fIdx];
    }
    for (int i = 0; i < NNINPUT_MAX_ONES; ++i) {
      int cIdx = onesIdx[NNINPUT_MAX_ONES * index + i];
      if (cIdx < nninputSize)
        output[nninputSize * index + cIdx] = 1.0;
    }
  }
}

hipError_t decompressNNInput(uint16_t* onesIdx, uint16_t* floatIdx, float* floatValue, float* output, int m, int nninputSize) {
  // 假设output已经分配并初始化为0

  // 设置线程块和网格大小
  int threadsPerBlock = 512;
  int blocksPerGrid = (m + threadsPerBlock - 1) / threadsPerBlock;

  hipMemset(output, 0, sizeof(float) * m * nninputSize);
  // 调用核函数
  decompressNNInputKernel << <blocksPerGrid, threadsPerBlock >> > (onesIdx, floatIdx, floatValue, output, m, nninputSize);

  // 等待GPU完成
  hipDeviceSynchronize();
  return hipGetLastError();
}
